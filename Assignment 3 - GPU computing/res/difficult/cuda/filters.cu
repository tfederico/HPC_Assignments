#include "hip/hip_runtime.h"
#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

const long nrThreads = 1000000;
const int filterHeight = 5;
const int filterWidth = 5;

__global__ void rgb2grayCUDA(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height) {
	unsigned int y = blockIdx.y;
	unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	float grayPix = 0.0f;
	float r = 0.0f;
	float g = 0.0f;
	float b = 0.0f;

	if ( x < width && y < height) {
		r = static_cast< float >(inputImage[(y * width) + x]);
		grayPix = (0.3f * r);
		g = static_cast< float >(inputImage[(width * height) + (y * width) + x]);
		grayPix	+= (0.59f * g);
		b = static_cast< float >(inputImage[(2 * width * height) + (y * width) + x]);
		grayPix += (0.11f * b);

		grayImage[(y * width) + x] = static_cast< unsigned char >(grayPix);
	}
}

__global__ void histogram1DCUDA(unsigned char *grayImage, const int width, const int height, unsigned int *histogram) {
	unsigned int y = blockIdx.y;
	unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(x < width && y < height){
		histogram[static_cast< unsigned int >(grayImage[(y * width) + x])] += 1;
	}
}

__global__ void contrast1DCUDA(unsigned char *grayImage, const int width, const int height, const int max, const int min){

	const float diff = max - min;

	unsigned int y = blockIdx.y;
	unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(x < width && y < height){
			unsigned char pixel = grayImage[(y * width) + x];

			if ( pixel < min ) {
				pixel = 0;
			}
			else if ( pixel > max ) {
				pixel = 255;
			}
			else {
				pixel = static_cast< unsigned char >(255.0f * (pixel - min) / diff);
			}

			grayImage[(y * width) + x] = pixel;
	}
}

__global__ void triangularSmoothCUDA(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height, float *filter) {
		unsigned int y = blockIdx.y;
		unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;

		if(x < width && y < height){
				unsigned int filterItem = 0;
				float filterSum = 0.0f;
				float smoothPix = 0.0f;

				for ( int fy = y - 2; fy < y + 3; fy++ ) {
					for ( int fx = x - 2; fx < x + 3; fx++ ) {

						if ( ((fy >= 0) && (fy < height)) && ((fx >= 0) && (fx < width)) ) {
							smoothPix += grayImage[(fy * width) + fx] * filter[filterItem];
							filterSum += filter[filterItem];
						}

						filterItem++;

					}
				}

				smoothPix /= filterSum;
				smoothImage[(y * width) + x] = static_cast< unsigned char >(smoothPix);
		}
}

void rgb2gray(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height, NSTimer &timer) {
		hipError_t devRetVal = hipSuccess;
		void *inputImage_d = 0;
		void *grayImage_d = 0;
		NSTimer kernelTime = NSTimer("kernelTime", false, false);
		NSTimer memoryTime = NSTimer("memoryTime", false, false);

		// Allocate device memory
		if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&inputImage_d), width * height * 3 * sizeof(unsigned char))) != hipSuccess ) {
			cerr << "Impossible to allocate device memory for inputImage_d." << endl;
			return;
		}
		if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&grayImage_d), width * height * sizeof(unsigned char))) != hipSuccess ) {
			cerr << "Impossible to allocate device memory for grayImage_d." << endl;
			return;
		}
		// Copy input to device
		memoryTime.start();
		if ( (devRetVal = hipMemcpy(inputImage_d, reinterpret_cast< void * >(inputImage), width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice)) != hipSuccess ) {
			cerr << "Impossible to copy inputImage_d to device." << endl;
			return;
		}
		memoryTime.stop();
		timer.stop();
		if ( (devRetVal = hipMemset(grayImage_d, 0, width * height * sizeof(unsigned char))) != hipSuccess ) {
			cerr << "Impossible to clean grayImage_d." << endl;
			return;
		}
		timer.start();


		dim3 gridSize = dim3(static_cast< unsigned int >(ceil(width / static_cast< float >(nrThreads))), height);
		dim3 blockSize = dim3(nrThreads);
		kernelTime.start();
		rgb2grayCUDA<<< gridSize, blockSize >>>(reinterpret_cast< unsigned char * >(inputImage_d), reinterpret_cast< unsigned char * >(grayImage_d), width, height);
		hipDeviceSynchronize();
		kernelTime.stop();

		// Copy back to host
		timer.stop();
		if ( (devRetVal = hipMemcpy(reinterpret_cast< void * >(grayImage), grayImage_d, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost)) != hipSuccess ) {
			cerr << "Impossible to copy grayImage_d to host." << endl;
			return;
		}
		hipFree(inputImage_d);
		hipFree(grayImage_d);
		timer.start();

		if(sizeof(grayImage) > sizeof(inputImage)){
			std::cerr << "Image too big!" << '\n';
		}
		cout << fixed << setprecision(6);
		cout << "rgb2gray (kernel): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
		cout << "rgb2gray (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;
}

void histogram1D(unsigned char *grayImage, unsigned char *histogramImage, const int width, const int height, unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, const unsigned int BAR_WIDTH, NSTimer &timer) {
		hipError_t devRetVal = hipSuccess;
		void *grayImage_d = 0;
		void *histogram_d = 0;
		NSTimer kernelTime = NSTimer("kernelTime", false, false);
		NSTimer memoryTime = NSTimer("memoryTime", false, false);

		// Allocate device memory
		if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&grayImage_d), width * height * sizeof(unsigned char))) != hipSuccess ) {
			cerr << "Impossible to allocate device memory for grayImage_d." << endl;
			return;
		}
		if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&histogram_d), HISTOGRAM_SIZE * sizeof(unsigned int))) != hipSuccess ) {
			cerr << "Impossible to allocate device memory for histogram_d." << endl;
			return;
		}

		// Copy input to device
		memoryTime.start();
		if ( (devRetVal = hipMemcpy(grayImage_d, reinterpret_cast< void * >(grayImage), width * height * sizeof(unsigned char), hipMemcpyHostToDevice)) != hipSuccess ) {
			cerr << "Impossible to copy grayImage_d to device." << endl;
			return;
		}
		if ( (devRetVal = hipMemcpy(histogram_d, reinterpret_cast< void * >(histogram), HISTOGRAM_SIZE * sizeof(unsigned int), hipMemcpyHostToDevice)) != hipSuccess ) {
			cerr << "Impossible to copy histogram_d to device." << endl;
			return;
		}
		memoryTime.stop();
		timer.stop();
		timer.start();

		dim3 gridSize = dim3(static_cast< unsigned int >(ceil(width / static_cast< float >(nrThreads))), height);
		dim3 blockSize = dim3(nrThreads);

		kernelTime.start();
		histogram1DCUDA<<< gridSize, blockSize >>>(reinterpret_cast< unsigned char * >(grayImage_d), width, height, reinterpret_cast< unsigned int * >(histogram_d));
		hipDeviceSynchronize();
		kernelTime.stop();

		// Copy back to host
		timer.stop();
		if ( (devRetVal = hipMemcpy(reinterpret_cast< void * >(histogram), histogram_d, HISTOGRAM_SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost)) != hipSuccess ) {
			cerr << "Impossible to copy histogram_d to host." << endl;
			return;
		}

		hipFree(grayImage_d);
		hipFree(histogram_d);
		timer.start();

		unsigned int max = 0;

		for ( unsigned int i = 0; i < HISTOGRAM_SIZE; i++ ) {
			if ( histogram[i] > max ) {
				max = histogram[i];
			}
		}

		for ( int x = 0; x < HISTOGRAM_SIZE * BAR_WIDTH; x += BAR_WIDTH ) {
			unsigned int value = HISTOGRAM_SIZE - ((histogram[x / BAR_WIDTH] * HISTOGRAM_SIZE) / max);
			for ( unsigned int y = 0; y < value; y++ ) {
				for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) {
					histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 0;
				}
			}
			for ( unsigned int y = value; y < HISTOGRAM_SIZE; y++ ) {
				for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) {
					histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 255;
				}
			}
		}

		cout << fixed << setprecision(6);
		cout << "histogram1D (kernel): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
		cout << "histogram1D (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;


}

void contrast1D(unsigned char *grayImage, const int width, const int height, unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, const unsigned int CONTRAST_THRESHOLD, NSTimer &timer) {
		hipError_t devRetVal = hipSuccess;
		void *grayImage_d = 0;
		NSTimer kernelTime = NSTimer("kernelTime", false, false);
		NSTimer memoryTime = NSTimer("memoryTime", false, false);

		// Allocate device memory
		if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&grayImage_d), width * height * sizeof(unsigned char))) != hipSuccess ) {
			cerr << "Impossible to allocate device memory for grayImage_d." << endl;
			return;
		}

		// Copy input to device
		memoryTime.start();
		if ( (devRetVal = hipMemcpy(grayImage_d, reinterpret_cast< void * >(grayImage), width * height * sizeof(unsigned char), hipMemcpyHostToDevice)) != hipSuccess ) {
			cerr << "Impossible to copy grayImage_d to device." << endl;
			return;
		}
		memoryTime.stop();
		timer.stop();
		timer.start();


		dim3 gridSize = dim3(static_cast< unsigned int >(ceil(width / static_cast< float >(nrThreads))), height);
		dim3 blockSize = dim3(nrThreads);

		unsigned int i = 0;

		while ( (i < HISTOGRAM_SIZE) && (histogram[i] < CONTRAST_THRESHOLD) ) {
			i++;
		}
		unsigned int min = i;

		i = HISTOGRAM_SIZE - 1;
		while ( (i > min) && (histogram[i] < CONTRAST_THRESHOLD) ) {
			i--;
		}
		unsigned int max = i;

		kernelTime.start();
		contrast1DCUDA<<< gridSize, blockSize >>>(reinterpret_cast< unsigned char * >(grayImage_d), width, height, max, min);
		hipDeviceSynchronize();
		kernelTime.stop();

		if ( (devRetVal = hipMemcpy(reinterpret_cast< void * >(grayImage), grayImage_d, height * width * sizeof(unsigned char), hipMemcpyDeviceToHost)) != hipSuccess ) {
			cerr << "Impossible to copy grayImage_d to host." << endl;
			return;
		}
		timer.stop();

		hipFree(grayImage_d);

		timer.start();

		cout << fixed << setprecision(6);
		cout << "contrast1D (kernel): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
		cout << "contrast1D (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;
}

void triangularSmooth(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height, float *filter, NSTimer &timer) {
		hipError_t devRetVal = hipSuccess;
		void *grayImage_d = 0;
		void *smoothImage_d = 0;
		void *filter_d = 0;
		const int filterSize = sizeof(filter);
		NSTimer kernelTime = NSTimer("kernelTime", false, false);
		NSTimer memoryTime = NSTimer("memoryTime", false, false);

		// Allocate device memory
		if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&grayImage_d), width * height * sizeof(unsigned char))) != hipSuccess ) {
			cerr << "Impossible to allocate device memory for grayImage_d." << endl;
			return;
		}
		if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&smoothImage_d), width * height * sizeof(unsigned char))) != hipSuccess ) {
			cerr << "Impossible to allocate device memory for smoothImage_d." << endl;
			return;
		}
		if ( (devRetVal = hipMalloc(reinterpret_cast<void ** >(&filter_d), filterHeight * filterWidth * sizeof(float))) != hipSuccess ) {
			cerr << "Impossible to allocate device memory for filter_d." << endl;
			return;
		}
		// Copy input to device
		memoryTime.start();
		if ( (devRetVal = hipMemcpy(grayImage_d, reinterpret_cast< void * >(grayImage), width * height * sizeof(unsigned char), hipMemcpyHostToDevice)) != hipSuccess ) {
			cerr << "Impossible to copy grayImage_d to device." << endl;
			return;
		}
		if ( (devRetVal = hipMemcpy(filter_d, reinterpret_cast<void * >(filter), filterHeight * filterWidth *  sizeof(float), hipMemcpyHostToDevice)) != hipSuccess ) {
			cerr << "Impossible to copy filter_d to device." << endl;
			return;
		}
		memoryTime.stop();
		timer.stop();
		if ( (devRetVal = hipMemset(smoothImage_d, 0, width * height * sizeof(unsigned char))) != hipSuccess ) {
			cerr << "Impossible to clean smoothImage_d." << endl;
			return;
		}
		timer.start();


		dim3 gridSize = dim3(static_cast< unsigned int >(ceil(width / static_cast< float >(nrThreads))), height);
		dim3 blockSize = dim3(nrThreads);
		kernelTime.start();
		triangularSmoothCUDA<<< gridSize, blockSize >>>(reinterpret_cast< unsigned char * >(grayImage_d), reinterpret_cast< unsigned char * >(smoothImage_d), width, height, reinterpret_cast<float * >(filter_d));
		hipDeviceSynchronize();
		kernelTime.stop();

		// Copy back to host
		timer.stop();
		if ( (devRetVal = hipMemcpy(reinterpret_cast< void * >(smoothImage), smoothImage_d, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost)) != hipSuccess ) {
			cerr << "Impossible to copy smoothImage_d to host." << endl;
			return;
		}

		hipFree(grayImage_d);
		hipFree(smoothImage_d);
		hipFree(filter_d);
		timer.start();

		cout << fixed << setprecision(6);
		cout << "triangularSmooth (kernel): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
		cout << "triangularSmooth (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

}
